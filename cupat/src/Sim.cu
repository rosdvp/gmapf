#include "hip/hip_runtime.h"
#include "../include/Sim.h"

#include <chrono>
#include <iostream>

#include "../include/misc/Cum.h"
#include "../include/misc/CuList.h"
#include "../include/AgentsMover.h"
#include "../include/PathAStarCPU.h"
#include "../include/PathFinder.h"
#include "../include/Helpers.h"

using namespace cupat;

void Sim::Init(const ConfigSim& config)
{
	printf("v0\n");

	_config = config;

	_agents = new Cum<CuList<Agent>>();
	_agents->HAllocAndMark(1, config.AgentsMaxCount);
}

void Sim::Destroy()
{
	std::cout << "[cupat] sim destroying.." << std::endl;

	CudaSyncAndCatch();

	delete _agentsMover;
	delete _pathFinder;

	_map->HFree();
	_map->DFree();
	delete _map;

	_agents->HFree();
	_agents->DFree();
	delete _agents;

	CudaCatch();

	CuDriverCatch(hipCtxDestroy(_cuContext));

	std::cout << "[cupat] sim destroyed" << std::endl;
}


void Sim::FillMap(const int* cells, float cellSize, int cellsCountX, int cellsCountY)
{
#ifndef CUPAT_NAV_MESH
	CuNodesMap::Desc desc;
	desc.CellsCountX = cellsCountX;
	desc.CellsCountY = cellsCountY;
	desc.Count = cellsCountX * cellsCountY;
	desc.CellSize = cellSize;

	_map = new Cum<CuNodesMap>();
	_map->HAllocAndMark(1, desc);
	auto hMap = _map->H(0);

	V2Int neibsDeltas[] = {
		{-1, 0},
		{1, 0},
		{0, -1},
		{0, 1},
		{1, 1},
		{1, -1},
		{-1, -1},
		{-1, 1}
	};
	for (int x = 0; x < cellsCountX; x++)
		for (int y = 0; y < cellsCountY; y++)
		{
			int nodeIdx = y * cellsCountX + x;
			if (cells[nodeIdx] == -1)
				continue;
			auto& node = hMap.At(nodeIdx);
			node.Val = cells[nodeIdx];

			V2Int cell(x, y);
			int counter = 0;
			for (auto& delta : neibsDeltas)
			{
				V2Int neib = cell + delta;
				if (neib.X < 0 || neib.X >= cellsCountX || neib.Y < 0 || neib.Y >= cellsCountY)
					continue;
				int neibNode = neib.Y * cellsCountX + neib.X;
				if (cells[neibNode] != -1)
					node.NeibsIdx[counter++] = neibNode;
			}
		}
#else
	throw std::exception();
#endif
}

void Sim::FillMap(const std::vector<CuNodesMap::Node>& nodes)
{
#ifdef CUPAT_NAV_MESH

	CuNodesMap::Desc desc;
	desc.Count = nodes.size();

	_map = new Cum<CuNodesMap>();
	_map->HAllocAndMark(1, desc);
	auto hMap = _map->H(0);

	for (int i = 0; i < nodes.size(); i++)
		hMap.At(i) = nodes[i];
#else
	throw std::exception();
#endif
}

int Sim::AddAgent(const V2Float& currPos)
{
	int currNodeIdx = -1;
	if (!_map->H(0).TryGetNodeIdx(currPos, &currNodeIdx))
		throw std::exception(("initial pos " + currPos.ToString() + " is invalid").c_str());

	Agent agent;
	agent.State = EAgentState::Idle;
	agent.CurrPos = currPos;
	agent.CurrNodeIdx = currNodeIdx;

	return _agents->H(0).Add(agent);
}

void Sim::SetAgentTargPos(int agentId, const V2Float& targPos)
{
	int targNodeIdx = -1;
	if (!_map->H(0).TryGetNodeIdx(targPos, &targNodeIdx))
		throw std::exception(("target pos " + targPos.ToString() + " is invalid").c_str());

	Agent& agent = _agents->H(0).At(agentId);
	agent.State = EAgentState::Search;
	agent.TargPos = targPos;
	agent.TargNodeIdx = targNodeIdx;
}

void Sim::DebugSetAgentPath(int agentId, const std::vector<V2Int>& path)
{
	Cum<CuList<V2Int>> cumPath;
	cumPath.HAllocAndMark(1, path.size());
	for (auto cell : path)
		cumPath.H(0).Add(cell);
	cumPath.CopyToDevice();

	Agent& agent = _agents->H(0).At(agentId);
	agent.State = EAgentState::Move;
	agent.Path = cumPath.DPtr(0);
	agent.PathStepIdx = -1;
}

void Sim::Start(bool isDebugSyncMode)
{
	hipSetDevice(0);
	CudaCatch();

	hipDevice_t device;
	CuDriverCatch(hipDeviceGet(&device, 0));
	CuDriverCatch(hipCtxCreate(&_cuContext, 0, device));
	CuDriverCatch(hipDeviceSetLimit(hipLimitMallocHeapSize, (size_t)1024 * (size_t)1024 * (size_t)128));

	//hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 32);
	//CudaCatch();

	CudaCatch();

	

	_map->CopyToDevice();
	CudaCatch();
	_agents->CopyToDevice();
	CudaCatch();

	_pathFinder = new PathFinder();
	_pathFinder->DebugSyncMode = isDebugSyncMode;
	_pathFinder->Init(
		*_map,
		*_agents,
		_config.PathFinderParallelAgents,
		_config.PathFinderThreadsPerAgents,
		_config.PathFinderQueueCapacity,
		_config.PathFinderHeuristicK
	);

	_agentsMover = new AgentsMover();
	_agentsMover->DebugSyncMode = isDebugSyncMode;
	_agentsMover->Init(
		*_map,
		*_agents,
		_config.AgentSpeed,
		_config.AgentRadius,
		_config.AgentsMaxCount
	);

	CuDriverCatch(hipCtxPopCurrent(nullptr));
}

void Sim::DoStep(float deltaTime)
{
	TIME_STAMP(tStart);

	CuDriverCatch(hipCtxPushCurrent(_cuContext));

	_pathFinder->AsyncPreFind();
	_agentsMover->AsyncPreMove();

	_pathFinder->Sync();
	_agentsMover->Sync();

	auto durPre = TIME_DIFF_MS(tStart);
	TIME_STAMP(tMain);

	_pathFinder->AsyncFind();
	_agentsMover->AsyncMove(deltaTime);

	_pathFinder->Sync();
	_agentsMover->Sync();

	auto durMain = TIME_DIFF_MS(tMain);
	TIME_STAMP(tPost);

	_pathFinder->PostFind();
	_agentsMover->PostMove();

	auto durPost = TIME_DIFF_MS(tPost);
	TIME_STAMP(tCopy);

	_agents->CopyToHost();

	auto durCopy = TIME_DIFF_MS(tCopy);

	CuDriverCatch(hipCtxPopCurrent(nullptr));

	auto durStep = TIME_DIFF_MS(tStart);

	TIME_APPLY_RECORD(durStep, _debugDurStepSum, _debugDurStepMax);
	TIME_APPLY_RECORD(durPre, _debugDurStepPreSum, _debugDurStepPreMax);
	TIME_APPLY_RECORD(durMain, _debugDurStepMainSum, _debugDurStepMainMax);
	TIME_APPLY_RECORD(durPost, _debugDurStepPostSum, _debugDurStepPostMax);
	TIME_APPLY_RECORD(durCopy, _debugDurStepCopySum, _debugDurStepCopyMax);
	_debugStepsCount += 1;
}

void Sim::DoStepOnlyFinder()
{
	TIME_STAMP(tStart);

	CuDriverCatch(hipCtxPushCurrent(_cuContext));

	_pathFinder->AsyncPreFind();
	_pathFinder->Sync();

	auto durPre = TIME_DIFF_MS(tStart);
	TIME_STAMP(tMain);

	_pathFinder->AsyncFind();
	_pathFinder->Sync();

	auto durMain = TIME_DIFF_MS(tMain);
	TIME_STAMP(tPost);

	_pathFinder->PostFind();

	auto durPost = TIME_DIFF_MS(tPost);

	CuDriverCatch(hipCtxPopCurrent(nullptr));

	auto durStep = TIME_DIFF_MS(tStart);

	TIME_APPLY_RECORD(durStep, _debugDurStepSum, _debugDurStepMax);
	TIME_APPLY_RECORD(durPre, _debugDurStepPreSum, _debugDurStepPreMax);
	TIME_APPLY_RECORD(durMain, _debugDurStepMainSum, _debugDurStepMainMax);
	TIME_APPLY_RECORD(durPost, _debugDurStepPostSum, _debugDurStepPostMax);
	_debugStepsCount += 1;
}

void Sim::DoStepOnlyMover(float deltaTime)
{
	TIME_STAMP(tStart);

	CuDriverCatch(hipCtxPushCurrent(_cuContext));

	_agentsMover->AsyncPreMove();
	_agentsMover->Sync();
	_agentsMover->AsyncMove(deltaTime);
	_agentsMover->Sync();
	_agentsMover->PostMove();

	_agents->CopyToHost();

	CuDriverCatch(hipCtxPopCurrent(nullptr));

	auto step = TIME_DIFF_MS(tStart);
	_debugDurStepSum += step;
	_debugDurStepMax = std::max(step, _debugDurStepMax);
	_debugStepsCount += 1;
}

const V2Float& Sim::GetAgentPos(int agentId)
{
	return _agents->H(0).At(agentId).CurrPos;
}

void Sim::DebugDump() const
{
	std::cout << "----------------------" << std::endl;

	TIME_STD_OUT("sim step", _debugDurStepSum, _debugDurStepMax, _debugStepsCount);
	TIME_STD_OUT("sim step pre", _debugDurStepPreSum, _debugDurStepPreMax, _debugStepsCount);
	TIME_STD_OUT("sim step main", _debugDurStepMainSum, _debugDurStepMainMax, _debugStepsCount);
	TIME_STD_OUT("sim step post", _debugDurStepPostSum, _debugDurStepPostMax, _debugStepsCount);
	TIME_STD_OUT("sim step copy", _debugDurStepCopySum, _debugDurStepCopyMax, _debugStepsCount);

	std::cout << std::endl;

	int count = _pathFinder->DebugRecordsCount;
	printf("path finder:\n");
	printf("prepare search ms, avg: %f max: %f, sum: %f\n", _pathFinder->DebugDurPrepareSearch / count, _pathFinder->DebugDurPrepareSearchMax,  _pathFinder->DebugDurPrepareSearch);
	printf("clear collections ms, avg: %f max: %f, sum: %f\n", _pathFinder->DebugDurClearCollections / count, _pathFinder->DebugDurClearCollectionsMax, _pathFinder->DebugDurClearCollections);
	printf("search ms, avg: %f max: %f, sum: %f\n", _pathFinder->DebugDurSearch / count, _pathFinder->DebugDurSearchMax, _pathFinder->DebugDurSearch);
	printf("build paths ms, avg: %f max: %f, sum: %f\n", _pathFinder->DebugDurBuildPaths / count, _pathFinder->DebugDurBuildPathsMax, _pathFinder->DebugDurBuildPaths);
	printf("attach paths ms, avg: %f max: %f, sum: %f\n", _pathFinder->DebugDurAttachPaths / count, _pathFinder->DebugDurAttachPathsMax,  _pathFinder->DebugDurAttachPaths);

	printf("\n");

	count = _agentsMover->DebugRecordsCount;
	printf("agents mover:\n");
	printf("find moving agents ms, avg: %f max: %f, sum: %f\n", _agentsMover->DebugDurFindAgents / count, _agentsMover->DebugDurFindAgentsMax, _agentsMover->DebugDurFindAgents);
	printf("move agents ms, avg: %f max: %f, sum: %f\n", _agentsMover->DebugDurMoveAgents / count, _agentsMover->DebugDurMoveAgentsMax, _agentsMover->DebugDurMoveAgents);
	printf("resolve collisions ms, avg: %f max: %f, sum: %f\n", _agentsMover->DebugDurResolveCollisions / count, _agentsMover->DebugDurResolveCollisionsMax, _agentsMover->DebugDurResolveCollisions);
	printf("update cells ms, avg: %f max: %f, sum: %f\n", _agentsMover->DebugDurUpdateCell / count, _agentsMover->DebugDurUpdateCellMax, _agentsMover->DebugDurUpdateCell);

	printf("----------------------\n");
}
