#include "../include/Sim.h"

#include <chrono>
#include <iostream>

#include "../include/Helpers.h"
#include "../include/kernels/KernelProcessAgent.h"
#include "../include/kernels/PathAStarCPU.h"
#include "../include/kernels/PathAStarMixed.h"

using namespace cupat;

Sim::~Sim()
{
	_map.HFree();
	_map.DFree();

	_agents.HFree();
	_agents.DFree();

	std::cout << "[cupat] sim destroyed" << std::endl;
}

void Sim::Init(const ConfigSim& config)
{
	_config = config;

	hipSetDevice(0);
	TryCatchCudaError("set device");
	hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 32);

	_map.HAllocAndMark(1, config.MapCountX, config.MapCountY);
	auto hMap = _map.H(0);
	for (int x = 0; x < config.MapCountX; x++)
		for (int y = 0; y < config.MapCountY; y++)
			hMap.At(x, y) = 0;

	_agents.HAllocAndMark(1, config.AgentsCount);
	auto hAgents = _agents.H(0);
	for (int i = 0; i < config.AgentsCount; i++)
		hAgents.Add({});
}

void Sim::SetAgentInitialPos(int agentId, const V2Float& currPos)
{
	Agent& agent = _agents.H(0).At(agentId);
	agent.CurrPos = currPos;
	agent.CurrCell = PosToCell(currPos);
}

void Sim::SetAgentTargPos(int agentId, const V2Float& targPos)
{
	Agent& agent = _agents.H(0).At(agentId);
	agent.TargPos = targPos;
	agent.TargCell = PosToCell(targPos);
	agent.IsNewPathRequested = true;
	agent.PathCellIdx = 0;
}

void Sim::SetObstacle(const V2Int& cell)
{
	_map.H(0).At(cell) = -1;
}

void Sim::Start()
{
	_map.CopyToDevice();
	TryCatchCudaError("allocate map");
	_agents.CopyToDevice();
	TryCatchCudaError("allocate agents");
}

bool Sim::DoStep(float deltaTime)
{
	CpuFindPathAStarInput cpuInp;
	cpuInp.Map = _map;
	cpuInp.Agents = _agents;
	cpuInp.AgentId = 0;

	int threadsCount = 1;

	FindPathAStarMixedInput mixedInp;
	mixedInp.Map = _map;
	mixedInp.Agents = _agents;
	mixedInp.AgentId = 0;

	auto time0 = std::chrono::high_resolution_clock::now();

	for (int i = 0; i < 1; i++)
	{
		//KernelFindPathAStar<<<1, 5>>>(inp);
		//KernelFindPathAStarMono<<<1, 1 >>>(inp);
		FindPathAStarMixed(mixedInp);
		//CpuFindPathAStar(cpuInp);
		//hipDeviceSynchronize();
	}

	auto time1 = std::chrono::high_resolution_clock::now();
	auto timeDelta = std::chrono::duration_cast<std::chrono::microseconds>(time1 - time0).count();
	std::cout << "time: " << timeDelta / 1000.0f << std::endl;

	return true;
}

const V2Float& Sim::GetAgentPos(int agentId)
{
	return _agents.H(0).At(agentId).CurrPos;
}

V2Int Sim::PosToCell(const V2Float& pos) const
{
	auto x = static_cast<int>(pos.X / _config.MapCellSize);
	auto y = static_cast<int>(pos.Y / _config.MapCellSize);
	return { x, y };
}